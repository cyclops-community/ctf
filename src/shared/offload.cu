#include "hip/hip_runtime.h"
/*Copyright (c) 2014, Edgar Solomonik, all rights reserved.*/

#include <complex>
#include <assert.h>
#include <stdio.h>
#include "int_timer.h"
#include <stdint.h>

#include "offload.h"
#include "../tensor/algstrct.h"
#include "../interface/timer.h"

#ifdef USE_CUDA
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include ""
#endif

namespace CTF_int{
  #ifndef PROFILE
  #ifndef CRITTER
  #define TAU_FSTART(ARG)
  #define TAU_FSTOP(ARG)
  #endif
  #define TAU_PROFILE(NAME,ARG,USER)
  #define TAU_PROFILE_TIMER(ARG1, ARG2, ARG3, ARG4)
  #define TAU_PROFILER_CREATE(ARG1, ARG2, ARG3, ARG4)
  #define TAU_PROFILE_STOP(ARG)
  #define TAU_PROFILE_START(ARG)
  #define TAU_PROFILE_SET_NODE(ARG)
  #define TAU_PROFILE_SET_CONTEXT(ARG)
  #endif

  #define ABORT                                   \
    do{                                           \
     assert(0); } while (0)

#ifdef USE_CUDA
  int initialized = 0;
  hipblasHandle_t cuhandle;

  void offload_init(){
    if (!initialized){
      int ndev=0;
      hipError_t err = hipGetDeviceCount(&ndev);
      assert(err == hipSuccess);
      assert(ndev > 0);
      hipblasStatus_t status = hipblasCreate(&cuhandle);
      assert(status == HIPBLAS_STATUS_SUCCESS);
    }
    initialized = 1;
  }

  void offload_exit(){
    if (initialized){
      hipblasStatus_t status = hipblasDestroy(cuhandle);
      assert(status == HIPBLAS_STATUS_SUCCESS);
      initialized = 0;
    }
  }

  offload_tsr::offload_tsr(algstrct const * sr_, int64_t size_) : offload_arr(size_*sr_->el_size) {
    sr = sr_;
    size = size_;
  }

  /*offload_tsr::~offload_tsr(){
  }*/

  LinModel<2> upload_mdl(upload_mdl_init,"upload_mdl");
  LinModel<2> download_mdl(download_mdl_init,"download_mdl");

  double estimate_download_time(int64_t size){
    double ps[] = {1.0, (double)size};
    return download_mdl.est_time(ps);
  }

  double estimate_upload_time(int64_t size){
    double ps[] = {1.0, (double)size};
    return upload_mdl.est_time(ps);
  }


  template <typename dtype>
  __global__ void gset_zero(dtype *arr, int64_t size, dtype val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i=idx; i<size; i+= gridDim.x*blockDim.x) {
      arr[i]=val;
    }
  }

  void offload_tsr::set_zero(){
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / (size);
    TAU_FSTART(set_zero);
    switch (sr->el_size){
      case 4:
        gset_zero<<<blockSize, numBlocks>>>((float*)dev_spr, size, ((float*)sr->addid())[0]);
        break;
      case 8:
        gset_zero<<<blockSize, numBlocks>>>((double*)dev_spr, size, ((double*)sr->addid())[0]);
        break;
      case 16:
        gset_zero<<<blockSize, numBlocks>>>((std::complex<double>*)dev_spr, size, ((std::complex<double>*)sr->addid())[0]);
        break;
      default:
        assert(0);
        break;
    }
    TAU_FSTOP(set_zero);
  }


  offload_arr::offload_arr(int64_t nbytes_){
    nbytes = nbytes_;
    TAU_FSTART(offload_malloc);
    hipError_t err = hipMalloc((void**)&dev_spr, nbytes);
    TAU_FSTOP(offload_malloc);
    assert(err == hipSuccess);
  }

  offload_arr::~offload_arr(){
    TAU_FSTART(offload_free);
    hipError_t err = hipFree(dev_spr);
    TAU_FSTOP(offload_free);
    assert(err == hipSuccess);
  }


  void offload_arr::download(char * host_spr){
     // not-quite-sure
    assert(initialized);
    TAU_FSTART(cuda_download);
    double st_time = MPI_Wtime();
    hipError_t err = hipMemcpy(host_spr, dev_spr, nbytes,
                                 hipMemcpyDeviceToHost);
    double exe_time = MPI_Wtime()-st_time;
    double tps[] = {exe_time, 1.0, (double)nbytes};
    download_mdl.observe(tps);
    TAU_FSTOP(cuda_download);
    assert(err == hipSuccess);
  }

  void offload_arr::upload(char const * host_spr){
     // not-quite-sure
    TAU_FSTART(cuda_upload);
    double st_time = MPI_Wtime();
    hipError_t err = hipMemcpy(dev_spr, host_spr, nbytes,
                                 hipMemcpyHostToDevice);

    double exe_time = MPI_Wtime()-st_time;
    double tps[] = {exe_time, 1.0, (double)nbytes};
    upload_mdl.observe(tps);
    TAU_FSTOP(cuda_upload);
    assert(err == hipSuccess);
  }



  void host_pinned_alloc(void ** ptr, int64_t size){
    TAU_FSTART(host_pinned_malloc);
    hipError_t err = hipHostAlloc(ptr, size, hipHostMallocMapped);
    TAU_FSTOP(host_pinned_malloc);
    assert(err == hipSuccess);
  }

  void host_pinned_free(void * ptr){
    TAU_FSTART(host_pinned_free);
    hipError_t err = hipHostFree(ptr);
    TAU_FSTOP(host_pinned_free);
    assert(err == hipSuccess);
  }
#endif

  template
  void offload_gemm(char          tA,
                    char          tB,
                    int           m,
                    int           n,
                    int           k,
                    double        alpha,
                    offload_tsr & A,
                    int           lda_A,
                    offload_tsr & B,
                    int           lda_B,
                    double        beta,
                    offload_tsr & C,
                    int           lda_C);

  template <>
  void offload_gemm<float>(char           tA,
                            char           tB,
                            int            m,
                            int            n,
                            int            k,
                            float         alpha,
                            float const * dev_A,
                            int            lda_A,
                            float const * dev_B,
                            int            lda_B,
                            float         beta,
                            float *       dev_C,
                            int            lda_C){
  #ifdef USE_CUDA
    assert(initialized);
  
    hipblasOperation_t cuA;  
    switch (tA){
      case 'n':
      case 'N':
        cuA = HIPBLAS_OP_N;
        break;
      case 't':
      case 'T':
        cuA = HIPBLAS_OP_T;
        break;
    }  
  
    hipblasOperation_t cuB;
    switch (tB){
      case 'n':
      case 'N':
        cuB = HIPBLAS_OP_N;
        break;
      case 't':
      case 'T':
        cuB = HIPBLAS_OP_T;
        break;
    }  
  
    hipblasStatus_t status = 
      hipblasDgemm(cuhandle, cuA, cuB, m, n, k, &alpha, 
                  dev_A, lda_A, 
                  dev_B, lda_B, &beta, 
                  dev_C, lda_C);
  #ifdef PROFILE
    hipDeviceSynchronize();
  #endif
    
    assert(status == HIPBLAS_STATUS_SUCCESS);
  #endif  
  }
  
  
  template <>
  void offload_gemm< std::complex<float> >(
                           char                         tA,
                           char                         tB,
                           int                          m,
                           int                          n,
                           int                          k,
                           std::complex<float>         alpha,
                           std::complex<float> const * dev_A,
                           int                          lda_A,
                           std::complex<float> const * dev_B,
                           int                          lda_B,
                           std::complex<float>         beta,
                           std::complex<float> *       dev_C,
                           int                          lda_C){
  #ifdef USE_CUDA
    assert(initialized);
    
    hipblasOperation_t cuA;  
    switch (tA){
      case 'n':
      case 'N':
        cuA = HIPBLAS_OP_N;
        break;
      case 't':
      case 'T':
        cuA = HIPBLAS_OP_T;
        break;
      case 'c':
      case 'C':
        cuA = HIPBLAS_OP_C;
        break;
    }  
  
    hipblasOperation_t cuB;
    switch (tB){
      case 'n':
      case 'N':
        cuB = HIPBLAS_OP_N;
        break;
      case 't':
      case 'T':
        cuB = HIPBLAS_OP_T;
        break;
      case 'c':
      case 'C':
        cuB = HIPBLAS_OP_C;
        break;
    }  
  
    TAU_FSTART(cublas_zgemm);
    hipblasStatus_t status = 
      hipblasZgemm(cuhandle, cuA, cuB, m, n, k, 
                  reinterpret_cast<hipDoubleComplex*>(&alpha), 
                  reinterpret_cast<const hipDoubleComplex*>(dev_A), lda_A, 
                  reinterpret_cast<const hipDoubleComplex*>(dev_B), lda_B, 
                  reinterpret_cast<hipDoubleComplex*>(&beta), 
                  reinterpret_cast<hipDoubleComplex*>(dev_C), lda_C);
  #ifdef PROFILE
    hipDeviceSynchronize();
  #endif
    TAU_FSTOP(cublas_zgemm);
    
    assert(status == HIPBLAS_STATUS_SUCCESS);
    assert(status == HIPBLAS_STATUS_SUCCESS);
  #endif
  }

  template <>
  void offload_gemm<double>(char           tA,
                            char           tB,
                            int            m,
                            int            n,
                            int            k,
                            double         alpha,
                            double const * dev_A,
                            int            lda_A,
                            double const * dev_B,
                            int            lda_B,
                            double         beta,
                            double *       dev_C,
                            int            lda_C){
  #ifdef USE_CUDA
    assert(initialized);

    hipblasOperation_t cuA;
    switch (tA){
      case 'n':
      case 'N':
        cuA = HIPBLAS_OP_N;
        break;
      case 't':
      case 'T':
        cuA = HIPBLAS_OP_T;
        break;
    }

    hipblasOperation_t cuB;
    switch (tB){
      case 'n':
      case 'N':
        cuB = HIPBLAS_OP_N;
        break;
      case 't':
      case 'T':
        cuB = HIPBLAS_OP_T;
        break;
    }

    hipblasStatus_t status =
      hipblasDgemm(cuhandle, cuA, cuB, m, n, k, &alpha,
                  dev_A, lda_A,
                  dev_B, lda_B, &beta,
                  dev_C, lda_C);
  #ifdef PROFILE
    hipDeviceSynchronize();
  #endif

    assert(status == HIPBLAS_STATUS_SUCCESS);
  #endif
  }


  template <>
  void offload_gemm< std::complex<double> >(
                           char                         tA,
                           char                         tB,
                           int                          m,
                           int                          n,
                           int                          k,
                           std::complex<double>         alpha,
                           std::complex<double> const * dev_A,
                           int                          lda_A,
                           std::complex<double> const * dev_B,
                           int                          lda_B,
                           std::complex<double>         beta,
                           std::complex<double> *       dev_C,
                           int                          lda_C){
  #ifdef USE_CUDA
    assert(initialized);

    hipblasOperation_t cuA;
    switch (tA){
      case 'n':
      case 'N':
        cuA = HIPBLAS_OP_N;
        break;
      case 't':
      case 'T':
        cuA = HIPBLAS_OP_T;
        break;
      case 'c':
      case 'C':
        cuA = HIPBLAS_OP_C;
        break;
    }

    hipblasOperation_t cuB;
    switch (tB){
      case 'n':
      case 'N':
        cuB = HIPBLAS_OP_N;
        break;
      case 't':
      case 'T':
        cuB = HIPBLAS_OP_T;
        break;
      case 'c':
      case 'C':
        cuB = HIPBLAS_OP_C;
        break;
    }

    TAU_FSTART(cublas_zgemm);
    hipblasStatus_t status =
      hipblasZgemm(cuhandle, cuA, cuB, m, n, k,
                  reinterpret_cast<hipDoubleComplex*>(&alpha),
                  reinterpret_cast<const hipDoubleComplex*>(dev_A), lda_A,
                  reinterpret_cast<const hipDoubleComplex*>(dev_B), lda_B,
                  reinterpret_cast<hipDoubleComplex*>(&beta),
                  reinterpret_cast<hipDoubleComplex*>(dev_C), lda_C);
  #ifdef PROFILE
    hipDeviceSynchronize();
  #endif
    TAU_FSTOP(cublas_zgemm);

    assert(status == HIPBLAS_STATUS_SUCCESS);
    assert(status == HIPBLAS_STATUS_SUCCESS);
  #endif
  }

  template <typename dtype>
  void offload_gemm(char           tA,
                    char           tB,
                    int            m,
                    int            n,
                    int            k,
                    dtype          alpha,
                    offload_tsr &  A,
                    int            lda_A,
                    offload_tsr &  B,
                    int            lda_B,
                    dtype          beta,
                    offload_tsr &  C,
                    int            lda_C){
    TAU_FSTART(cuda_gemm);
    offload_gemm(tA, tB, m, n, k, alpha, (dtype*)A.dev_spr, lda_A, (dtype*)B.dev_spr, lda_B, beta, (dtype*)C.dev_spr, lda_C);
    TAU_FSTOP(cuda_gemm);
  }
}
